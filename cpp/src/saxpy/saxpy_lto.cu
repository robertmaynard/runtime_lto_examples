/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <random>
#include <vector>

#include "hip/hip_runtime.h"
#include "cuda_wrapper.hpp"
#include "grid_1d.hpp"
#include "saxpy_setup.h"

#include <cub/detail/launcher/cuda_driver.cuh>


hipLibrary_t load_fatbins(hipDevice_t, std::vector<std::string>);

void run_saxpy(hipDevice_t cuda_device,
               hipcub::detail::CudaDriverLauncher& launcher,
               std::string const& algorithm_text,
               std::vector<std::string> const& fatbins,
               saxpy_memory& saxpy) {
  const auto n = static_cast<std::int64_t>(saxpy.x->size());

  std::cout << "Start loading " << algorithm_text << " LTO FATBINS \n";
  auto cuda_lib = load_fatbins(cuda_device, fatbins);
  std::cout << "Finished loading \n";
  // Get kernel pointer out of the library
  hipKernel_t kernel;
  std::cout << "Launch " << algorithm_text << "  with " << n << " elements\n";
  DEMO_CUDA_TRY(hipLibraryGetKernel(&kernel, cuda_lib, "saxpy"));
  launcher.doit(kernel, saxpy.x->begin(), saxpy.y->begin(), n);
}

int main(int, char**) {

  hipDevice_t cuda_device;
  hipCtx_t cuda_context;
  hipInit(0);
  DEMO_CUDA_TRY(hipDeviceGet(&cuda_device, 0));
  DEMO_CUDA_TRY(hipCtxCreate(&cuda_context, 0, cuda_device));

  rmm::hip_stream stream{};
  saxpy_memory saxpy{stream};

  // Build up a launcher for kernels with the same grid, block, etc
  const auto n = static_cast<std::int64_t>(saxpy.x->size());

  common::grid_1d grid_config{n, common::block_size};
  const auto grid_size = static_cast<std::uint32_t>(grid_config.num_blocks);
  constexpr size_t shared_mem = 0;
  hipcub::detail::CudaDriverLauncher launcher{
    dim3{grid_size, 1, 1},
    dim3{common::block_size, 1, 1},
    shared_mem,
    stream.value()};

  auto fast_saxpy_fatbins = std::vector<std::string>{
      "saxpy_compute.fatbin", "saxpy_grid_stride.fatbin"};
  run_saxpy(cuda_device, launcher, "fast saxpy", fast_saxpy_fatbins, saxpy);

  auto slow_saxpy_fatbins_1 = std::vector<std::string>{
      "saxpy_compute_slow_1.fatbin", "saxpy_grid_stride.fatbin"};
  run_saxpy(cuda_device, launcher, "slow saxpy pass 1", slow_saxpy_fatbins_1, saxpy);

  auto slow_saxpy_fatbins_2 = std::vector<std::string>{
      "saxpy_compute_slow_2.fatbin", "saxpy_grid_stride.fatbin"};
  run_saxpy(cuda_device, launcher, "slow saxpy pass 2", slow_saxpy_fatbins_2, saxpy);

  std::vector<float> host_y;
  host_y.resize(n);

  hipMemcpyAsync(host_y.data(), saxpy.y->begin(), n * sizeof(float),
                  hipMemcpyDefault, stream.value());

  hipStreamSynchronize(stream.value());

  return 0;
}
