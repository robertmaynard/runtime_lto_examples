#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "LaunchKernelEntry.h"

#include <cstring>
#include <iostream>


namespace {
  std::string make_launch_key(std::vector<std::string> const& params) {
    std::string k{};
    for(auto& p : params) {
      k += p + "_";
    }
    return k;
  }

  // We can make a better RAII wrapper around nvjitlinkhandle
  void check_nvjitlink_result(nvJitLinkHandle handle, nvJitLinkResult result) {
    if (result != NVJITLINK_SUCCESS) {
      std::cerr << "\n nvJITLink failed with error " << result << '\n';
      size_t log_size = 0;
      result = nvJitLinkGetErrorLogSize(handle, &log_size);
      if (result == NVJITLINK_SUCCESS && log_size > 0) {
        std::unique_ptr<char[]> log{new char[log_size]};
        result = nvJitLinkGetErrorLog(handle, log.get());
        if (result == NVJITLINK_SUCCESS) {
          std::cerr << "nvJITLink error log: " << log.get() << '\n';
        }
      }
      exit(1);
    }
  }
}

LaunchKernelEntry::LaunchKernelEntry(std::vector<std::string> const& params)
    : launch_arg_count(params.size()),
    launch_key(make_launch_key(params)){}

std::size_t LaunchKernelEntryHash::operator()(
    std::vector<std::string> const& params) const noexcept {
    return std::hash<std::string>{}(make_launch_key(params));
}

bool LaunchKernelEntryEqual::operator()(
    std::vector<std::string> const& params,
    std::unique_ptr<LaunchKernelEntry> const& entry) const noexcept {

    if( params.size() == entry->launch_arg_count ) {
      auto key = make_launch_key(params);
      return entry->launch_key == key;
    }
    return false;
}

FatbinLaunchKernelEntry::FatbinLaunchKernelEntry(
    std::vector<std::string> const& params, unsigned char const* view)
    : LaunchKernelEntry(params),
      data_size(std::strlen((char*)view)),
      data_view(view) {}

bool FatbinLaunchKernelEntry::add_to(nvJitLinkHandle& handle) const {
    auto result =
        nvJitLinkAddData(handle, NVJITLINK_INPUT_FATBIN, this->data_view,
                         this->data_size, this->launch_key.c_str());
    check_nvjitlink_result(handle, result);
    return true;
}

NVRTCLaunchKernelEntry::NVRTCLaunchKernelEntry(
    std::vector<std::string> const& params, std::string const& mname,
    std::size_t size,
    std::unique_ptr<char[]>&& p)
    : LaunchKernelEntry(params),
      mangled_name(mname),
      data_size(size),
      program(std::move(p)) {}

bool NVRTCLaunchKernelEntry::add_to(nvJitLinkHandle& handle) const {
    auto result =
        nvJitLinkAddData(handle, NVJITLINK_INPUT_LTOIR, this->program.get(),
                         this->data_size, this->launch_key.c_str());
    check_nvjitlink_result(handle, result);

    return true;
}
