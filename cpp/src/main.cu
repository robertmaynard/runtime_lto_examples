/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <memory>
#include <vector>

#include "cuda_wrapper.hpp"

#include "hip/hip_runtime.h"

#include <cub/detail/launcher/cuda_driver.cuh>

hipLibrary_t load_fatbins(hipDevice_t, std::vector<std::string>);

// NOTICES:
// When converting this to production code we need to use a
// dlopen wrapper around cuda driver so that we can gracefully fail
// at runtime

int main() {

  hipDevice_t cuda_device;
  hipCtx_t cuda_context;
  hipInit(0);
  DEMO_CUDA_TRY(hipDeviceGet(&cuda_device, 0));
  DEMO_CUDA_TRY(hipCtxCreate(&cuda_context, 0, cuda_device));

  std::cout << "Started Loading LTO FATBINS \n";
  auto cuda_lib = load_fatbins(
    cuda_device,
    std::vector<std::string>{"kernels.fatbin"});
  std::cout << "Finished Loading LTO FATBINS \n";

  //Build up a launcher for kernels with the same grid, block, etc
  constexpr dim3 grid = {1, 1, 1};
  constexpr dim3 block = {1, 1, 1};
  constexpr size_t shared_mem = 0;
  hipStream_t stream;
  DEMO_CUDA_TRY(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  hipcub::detail::CudaDriverLauncher launcher{grid, block, shared_mem, stream};

  // Get kernel pointer out of the library
  hipKernel_t kernel;
  std::cout << "Launch hello world \n";
  DEMO_CUDA_TRY(hipLibraryGetKernel(&kernel, cuda_lib, "hello_world"));
  launcher.doit(kernel);

  DEMO_CUDA_TRY(hipStreamSynchronize(stream));


  DEMO_CUDA_TRY(hipStreamDestroy(stream));
  DEMO_CUDA_TRY(hipLibraryUnload(cuda_lib));
  DEMO_CUDA_TRY(hipCtxDestroy(cuda_context));
  return 0;
}
